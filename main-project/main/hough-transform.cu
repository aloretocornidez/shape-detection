#include "hip/hip_runtime.h"
#include "hough-transform.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hough_transform_kernel_naive(char input[], char mask[], char output[], int rows, int cols, int maskWidth)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int pixVal = 0;

    if (row < rows && col < cols)
    {
        int startCol = col - maskWidth / 2;
        int startRow = row - maskWidth / 2;

        for (int j = 0; j < maskWidth; j++)
        {
            for (int k = 0; k < maskWidth; k++)
            {
                int curRow = startRow + j;
                int curCol = startCol + k;

                if (curRow > -1 && curRow < rows && curCol > -1 && curCol < cols)
                {
                    pixVal += input[curRow * cols + curCol] * mask[j * maskWidth + k];
                }
            }
        }
    }

    output[row * cols + col] = pixVal;
}

__global__ void add_kernel_basic(int size, int *input1, int *input2)
{

    int thread = threadIdx.x + blockIdx.x * blockDim.x;

    if (thread < size)
    {
        input1[thread] = input1[thread] + input2[thread];
    }
}

void cudaAddKernel(int array_size, int *array_1, int *array_2)
{

    std::cout << "Running Kernel Wrapper" << std::endl;

    // Initializing pointers to the gpu memory
    int *gpu_array_1;
    int *gput_array_2;

    // allocate memory on device, check for failure
    if (hipMalloc((void **)&gpu_array_1, array_size * sizeof(int)) != hipSuccess)
    {
        std::cout << "malloc error for gpuInput1" << std::endl;
    }
    if (hipMalloc((void **)&gput_array_2, array_size * sizeof(int)) != hipSuccess)
    {
        std::cout << "malloc error for gpuInput2" << std::endl;
    }

    // copy data to device, check for failure, free device if needed

    hipError_t err; // Use this whenever calling hipMalloc and hipMemcpy.

    err = hipMemcpy(gpu_array_1, array_1, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(gput_array_2, array_2, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    dim3 mygrid(ceil(array_size / 256.0));
    dim3 myblock(256);

    add_kernel_basic<<<mygrid, myblock>>>(array_size, gpu_array_1, gput_array_2);

    // copy data to host, check for failure, free device if needed
    if (hipMemcpy(array_1, gpu_array_1, array_size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(gpu_array_1);
        hipFree(gput_array_2);
        printf("data transfer error from device to host on input1\n");
    }
    if (hipMemcpy(array_2, gput_array_2, array_size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(gpu_array_1);
        hipFree(gput_array_2);
        printf("data transfer error from device to host on input2\n");
    }

    std::cout << "Finished Kernel Wrapper execution" << std::endl;
}