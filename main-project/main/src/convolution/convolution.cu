
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<time.h>
#include<stdio.h>
//#include <cuda_runtime.h>
#include <iostream>
#include <assert.h>
#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

void serialConvolution(char input[], char mask[], char output[], int rows, int cols, int maskWidth);

//NAIVE 2D CONVOLUTION
__global__ void naiveConvolution(char input[], char mask[], char output[], int rows, int cols, int maskWidth){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int pixVal = 0;

    if(row < rows && col < cols){
        int startCol = col - maskWidth / 2;
        int startRow = row - maskWidth / 2;

        for(int j = 0; j < maskWidth; j++){
            for(int k = 0; k < maskWidth; k++){
                int curRow = startRow + j;
                int curCol = startCol + k;

                if(curRow > -1 && curRow < rows && curCol > -1 && curCol < cols){
                    pixVal += input[curRow * cols + curCol] * mask[j * maskWidth + k];
                }
            }
        }
    }

    output[row * cols + col] = pixVal;
    
}

//MAIN FUNCTION
int main(void){
    srand(time(NULL)); //Set up random values
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //We will compute a convolution image on a 720p made up image
    char inputImage[720][1280];
    char outputImage[720][1280];
    char mask3x3[3][3];
    char mask5x5[5][5];

    //Initialize input image 
    for(int i = 0; i < 720; i++){
        for(int j = 0; j < 1280; j++){
            inputImage[i][j] = rand() % 10;
        }
    }

    //Initialize 3x3 mask and 5x5 mask
    for(int i = 0; i < 3; i++){
        for(int j = 0; j < 3; j++){
            mask3x3[i][j] = rand() % 10;
        }
    }
    for(int i = 0; i < 5; i++){
        for(int j = 0; j < 5; j++){
            mask5x5[i][j] = rand() % 10;
        }
    }

    //Apply convolution kernel to test values for accuracy
    //Will be using a 5x5 for now
    int kCols = 5;
    int kRows = 5;
    int kColDisplace = kCols / 2;
    int kRowDisplace = kRows / 2;

	std::cout << "Hello!" << std::endl;

    hipEventRecord(start);
    for(int i=0; i < 720; ++i)              // rows
    {
        for(int j=0; j < 1280; ++j)          // columns
        {
            int startRow = i - kRowDisplace;
            int startCol = j - kColDisplace;
            char sum = 0;
            
            for(int m=0; m < kRows; ++m) { //Kernel rows
                for(int n=0; n < kCols; ++n) { //Kernel Cols
                    //int nn = kCols - 1 - n;  // column index of flipped kernel
                    int currRow = startRow + m;
                    int currCol = startCol + n;

                    if(currRow > -1 && currRow < 720 && currCol > -1 && currCol < 1280){
                        sum += inputImage[currRow][currCol] * mask5x5[m][n];
                    }
                }
            }
            outputImage[i][j] = sum;
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Elapsed time for serial: " << milliseconds << std::endl;

    //Now we will do a naive implementation on CUDA
    char* hostInput;
    char* hostMask;
    char* hostOutput;
    char* deviceInput;
    char* deviceMask;
    char* deviceOutput;
    int rows = 720;
    int cols = 1280;
    int maskVal = 5;
    hipEvent_t startNaive, stopNaive;
    hipEventCreate(&startNaive);
    hipEventCreate(&stopNaive);

    //Allocate Memory on host side
    hostInput = (char*)malloc(rows * cols * sizeof(char));
    hostMask = (char*)malloc(maskVal * maskVal * sizeof(char));
    hostOutput = (char*)malloc(rows * cols * sizeof(char));

    //Populare arrays on the host side
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < cols; j++){
            hostInput[i * rows + j] = inputImage[i][j];
        }
    }
    for(int i = 0; i < maskVal; i++){
        for(int j = 0; j < maskVal; j++){
            hostInput[i * rows + j] = mask5x5[i][j];
        }
    }

    //Allocate GPU memory here
    CUDA_CHECK(
        hipMalloc((void **)&deviceInput, rows * cols * sizeof(char)));
    CUDA_CHECK(
        hipMalloc((void **)&deviceMask, maskVal * maskVal * sizeof(char)));
    CUDA_CHECK(
        hipMalloc((void **)&deviceOutput, rows * cols * sizeof(char)));
    CUDA_CHECK(hipDeviceSynchronize());

    //Populate arrays on device side
    CUDA_CHECK(hipMemcpy(deviceInput, hostInput,
                            rows * cols * sizeof(char),
                            hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceMask, hostMask,
                            maskVal * maskVal * sizeof(char),
                            hipMemcpyHostToDevice));
    CUDA_CHECK(hipDeviceSynchronize());

    //Call the naive kernel
    hipEventRecord(startNaive);
    int blockSize = 32;
    dim3 blockDim(blockSize,blockSize), gridDim( 1 + (cols - 1) / blockSize,
                                             1 + (rows - 1) / blockSize);
    naiveConvolution<<<gridDim, blockDim>>>
        (deviceInput, deviceMask, deviceOutput, rows, cols, maskVal);
    hipEventRecord(stopNaive);
    hipEventSynchronize(stopNaive);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startNaive, stopNaive);
    std::cout << "Elapsed time for naive: " << milliseconds <<std::endl;
	hipDeviceSynchronize();

    CUDA_CHECK(hipMemcpy(hostOutput, deviceOutput,
                            rows * cols * sizeof(char),
                            hipMemcpyDeviceToHost));
    
    //Check wether output is correct
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < cols; j++){
            assert(hostOutput[i * rows + j] == outputImage[i][j]);
        }
    }
    

    //


    //Memory Freeing
    //CUDA
    hipFree(deviceInput);
    hipFree(deviceMask);
    hipFree(deviceOutput);

    //CPU
    free(hostInput);
    free(hostMask);
    free(hostOutput);
}