#include "hip/hip_runtime.h"
#include "hough-transform.hpp"
#include <iostream>
#include <opencv2/core.hpp>

__global__ void hough_transform_kernel_naive(uchar *inputImage, int height, int width)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int column = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < height && column < width)
    {

        inputImage[row * width + column] = 0;
    }
}

void cudaHoughTransform(cv::Mat grayscaleInputImage, cv::InputArray circles)
{

    std::cout << "Running CUDA hough transform." << std::endl;

    int imageHeight = grayscaleInputImage.rows;
    int imageWidth = grayscaleInputImage.cols;

    // Initializing pointers for the GPU memory
    uchar *gpuImageBuffer;

    // Error detection
    hipError_t err;

    // Allocate GPU Memory
    err = hipMalloc((void **)&gpuImageBuffer, imageHeight * imageWidth * sizeof(uchar));
    if (err != hipSuccess)
    {
        std::cout << "malloc error for gpuInput1" << std::endl;
    }

    // Copy Data from host to Device
    err = hipMemcpy(gpuImageBuffer, &grayscaleInputImage.at<uchar>(0, 0), imageHeight * imageWidth * sizeof(uchar), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    std::cout << "Cuda Memory Allocated.\n" << std::endl;



    // Execute kernel
    dim3 mygrid(ceil(imageHeight * imageHeight / 256.0));
    dim3 myblock(256);

    std::cout << "Executing kernel." << std::endl;
    hough_transform_kernel_naive<<<mygrid, myblock>>>(gpuImageBuffer, imageHeight, imageWidth);
    std::cout << "Kernel execution complete.\n" << std::endl;

    // Copy data from device to host
    // copy data to host, check for failure, free device if needed
    std::cout << "Copying from device to host." << std::endl;
    if (hipMemcpy(&grayscaleInputImage.at<uchar>(0, 0), gpuImageBuffer, imageHeight * imageWidth * sizeof(uchar), hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(gpuImageBuffer);
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    std::cout << "Memory Copied." << std::endl;

    // Free allocated memory
    hipFree(gpuImageBuffer);


    std::cout << "Finished CUDA hough transform.\n" << std::endl;
}

__global__ void add_kernel_basic(int size, int *input1, int *input2)
{

    int thread = threadIdx.x + blockIdx.x * blockDim.x;

    if (thread < size)
    {
        input1[thread] = input1[thread] + input2[thread];
    }
}

void cudaAddKernel(int array_size, int *array_1, int *array_2)
{

    std::cout << "Running Kernel Wrapper" << std::endl;

    // Initializing pointers to the gpu memory
    int *gpu_array_1;
    int *gput_array_2;

    // allocate memory on device, check for failure
    if (hipMalloc((void **)&gpu_array_1, array_size * sizeof(int)) != hipSuccess)
    {
        std::cout << "malloc error for gpuInput1" << std::endl;
    }
    if (hipMalloc((void **)&gput_array_2, array_size * sizeof(int)) != hipSuccess)
    {
        std::cout << "malloc error for gpuInput2" << std::endl;
    }

    // copy data to device, check for failure, free device if needed

    hipError_t err; // Use this whenever calling hipMalloc and hipMemcpy.

    err = hipMemcpy(gpu_array_1, array_1, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(gput_array_2, array_2, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Execute kernel.
    dim3 mygrid(ceil(array_size / 256.0));
    dim3 myblock(256);

    add_kernel_basic<<<mygrid, myblock>>>(array_size, gpu_array_1, gput_array_2);

    // copy data to host, check for failure, free device if needed
    if (hipMemcpy(array_1, gpu_array_1, array_size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(gpu_array_1);
        hipFree(gput_array_2);
        printf("data transfer error from device to host on input1\n");
    }
    if (hipMemcpy(array_2, gput_array_2, array_size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(gpu_array_1);
        hipFree(gput_array_2);
        printf("data transfer error from device to host on input2\n");
    }

    hipFree(gpu_array_1);
    hipFree(gput_array_2);

    std::cout << "Finished Kernel Wrapper execution" << std::endl;
}
