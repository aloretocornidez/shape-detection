#include "hip/hip_runtime.h"
#include "hough-transform.hpp"
#include <iostream>
#include <opencv2/opencv.hpp>

__global__ void hough_transform_kernel_naive(uchar *inputImage, int height, int width)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int column = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < height && column < width)
    {
        // printf("%d\n", inputImage[row * width + column]);

        inputImage[row * width + column] = 128;

        // printf("%d\n", inputImage[row * width + column]);
    }

}

void cudaHoughTransform(cv::Mat &grayscaleInputImage, cv::InputArray circles)
{

    std::cout << "Running CUDA hough transform." << std::endl;

    int imageRows = grayscaleInputImage.rows;
    int imageColumns = grayscaleInputImage.cols;


    // Initializing pointers for the GPU memory
    uchar *gpuImageBuffer;

    // Error detection
    hipError_t err;

    // Allocate GPU Memory
    err = hipMalloc((void **)&gpuImageBuffer, imageRows * imageColumns * sizeof(uchar));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }



    // Copy Data from host to Device
    err = hipMemcpy(gpuImageBuffer, grayscaleInputImage.ptr<uchar>(0, 0), imageRows * imageColumns * sizeof(uchar), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    // std::cout << "Cuda Memory Allocated." << std::endl;

    // Execute kernel
    const int threads = 32;
    dim3 mygrid(ceil(imageColumns / (threads * 1.0)), ceil(imageRows / (threads * 1.0)));
    dim3 myblock(threads, threads);

    std::cout << "Executing kernel." << std::endl;
    hough_transform_kernel_naive<<<mygrid, myblock>>>(gpuImageBuffer, imageRows, imageColumns);
    std::cout << "Kernel execution complete." << std::endl;

    // Copy data from device to host
    std::cout << "Copying from device to host." << std::endl;
    err = hipMemcpy(grayscaleInputImage.ptr<uchar>(0, 0), gpuImageBuffer, imageRows * imageColumns * sizeof(uchar), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        hipFree(gpuImageBuffer);
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    std::cout << "Memory Copied." << std::endl;

    hipDeviceSynchronize();

    // for (int row = 0; row < grayscaleInputImage.rows; row++)
    // {
    //     for (int column = 0; column < grayscaleInputImage.cols; column++)
    //     {
    //         std::cout << (int)grayscaleInputImage.at<uchar>(row, column) << std::endl;
    //         // grayscaleInputImage.at<uchar>(row, column) = 128; // column % 256;
    //     }
    // }

    // Free allocated memory
    hipFree(gpuImageBuffer);



    std::cout << "Finished CUDA hough transform." << std::endl;
}

__global__ void add_kernel_basic(int size, int *input1, int *input2)
{

    int thread = threadIdx.x + blockIdx.x * blockDim.x;

    if (thread < size)
    {
        input1[thread] = input1[thread] + input2[thread];
    }
}

void cudaAddKernel(int array_size, int *array_1, int *array_2)
{

    std::cout << "Running Kernel Wrapper" << std::endl;

    // Initializing pointers to the gpu memory
    int *gpu_array_1;
    int *gput_array_2;

    // allocate memory on device, check for failure
    if (hipMalloc((void **)&gpu_array_1, array_size * sizeof(int)) != hipSuccess)
    {
        std::cout << "malloc error for gpuInput1" << std::endl;
    }
    if (hipMalloc((void **)&gput_array_2, array_size * sizeof(int)) != hipSuccess)
    {
        std::cout << "malloc error for gpuInput2" << std::endl;
    }

    // copy data to device, check for failure, free device if needed

    hipError_t err; // Use this whenever calling hipMalloc and hipMemcpy.

    err = hipMemcpy(gpu_array_1, array_1, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(gput_array_2, array_2, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Execute kernel.
    dim3 mygrid(ceil(array_size / 256.0));
    dim3 myblock(256);

    add_kernel_basic<<<mygrid, myblock>>>(array_size, gpu_array_1, gput_array_2);

    // copy data to host, check for failure, free device if needed
    if (hipMemcpy(array_1, gpu_array_1, array_size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(gpu_array_1);
        hipFree(gput_array_2);
        printf("data transfer error from device to host on input1\n");
    }
    if (hipMemcpy(array_2, gput_array_2, array_size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(gpu_array_1);
        hipFree(gput_array_2);
        printf("data transfer error from device to host on input2\n");
    }

    hipFree(gpu_array_1);
    hipFree(gput_array_2);

    std::cout << "Finished Kernel Wrapper execution" << std::endl;
}
