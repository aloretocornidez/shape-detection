#include "hip/hip_runtime.h"
#include "hough-transform.hpp"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

// Thread Block and Memory Allocation Parameters.
#define BLOCK_DIMENSIONS 32

// Hough Tranform Kernel Definitions
#define HOUGH_TRANSFORM_NAIVE_KERNEL 1
#define HOUGH_TRANSFORM_NAIVE_KERNEL2 2
#define HOUGH_TRANSFORM_SHARED_LOCAL_ACCUMULATOR 3
#define DEGREES_TO_RADIANS 0.0120830485



void cpuKernelHoughTransform(cv::Mat &srcImage, std::vector<cv::Vec3f> &srcCircles, int distance, int minimumRadius, int maximumRadius)
{
    std::cout << "Executing the Hough Transform on the CPU." << std::endl;

    if (minimumRadius < 0)
    {
        std::cerr << "Minimum radius must be 1 or greater." << std::endl;
        exit(-1);
    }
    if (minimumRadius == 0)
    {
        minimumRadius = 5;
    }
    if (maximumRadius == 0)
    {
        maximumRadius = std::min(srcImage.rows - 1, srcImage.cols - 1) / 2;
    }

    /* Begin Algoritm */
    for (int radius = minimumRadius; radius < maximumRadius; radius++)
    {
        // Note, the threshold for the number of pixels is dynamically set.
        int threshold = ((log(radius * 2 / 3)) * 80) / log(3);

        for (int row = radius; row < srcImage.rows - radius; row += distance)
        {
            for (int column = radius; column < srcImage.cols - radius; column += distance)
            {
                int accumulator = 0;

                // Check if the a circle exists at the coordinate point (with the current radius).
                for (int theta = 0; theta < 360; theta++)
                {
                    // Checking all 4 cardinal directions.
                    int x;
                    int y;
                    int deltaX = cos(theta * DEGREES_TO_RADIANS) * radius;
                    int deltaY = sin(theta * DEGREES_TO_RADIANS) * radius;
                    x = deltaX + column;
                    y = deltaY + row;

                    if (srcImage.at<uchar>(x, y) < 10)
                    {
                        accumulator++;
                    }
                }

                // Adding the coordinate if the contained enough edge pixels.
                if (accumulator > threshold)
                {
                    srcCircles.push_back({(float)row, (float)column, (float)radius});
                }
            }
        }
    }

    std::cout << "Execution of the Hough Transform on the CPU completed." << std::endl;
}

// This kernel uses the global memory to write to the R-table.
// Each thread will test
__global__ void hough_transform_kernel_naive(uchar *srcImage, unsigned int *rTable, int imageRows, int imageColumns, int minimumRadius, int maximumRadius, int distance)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int column = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if the thread is within image bounds.
    if (row < (imageRows - minimumRadius) && row > minimumRadius && column < (imageColumns - minimumRadius) && column > minimumRadius)
    {

        for (int radius = minimumRadius; radius < maximumRadius - minimumRadius; radius++)
        {
            int threshold = ((log10f(radius * 2 / 3)) * 80) / log10f(3);

            for (int theta = 0; theta < 360; theta++)
            {
                int deltaX = cos(DEGREES_TO_RADIANS * theta) * radius;
                int deltaY = sin(DEGREES_TO_RADIANS * theta) * radius;

                int imageIndex = (row + deltaY) * imageColumns + (column + deltaX);

                int pixelValue = srcImage[imageIndex];
                if (pixelValue < 10)
                {
                    atomicAdd(&rTable[(radius * imageColumns * imageRows) + row * imageColumns + column], 1);
                }
            }
        }
    }
}

__global__ void hough_transform_kernel_naive2(uchar *srcImage, unsigned int *rTable, int imageRows, int imageColumns, int minimumRadius, int maximumRadius, int distance)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int column = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if the thread is within image bounds.
    if (row < (imageRows - minimumRadius) && row > minimumRadius && column < (imageColumns - minimumRadius) && column > minimumRadius)
    {

        for (int radius = minimumRadius; radius < maximumRadius - minimumRadius; radius++)
        {
            int threshold = ((log10f(radius * 2 / 3)) * 80) / log10f(3);
            int accumulator = 0;

            for (int theta = 0; theta < 360; theta++)
            {
                int deltaX = cos(DEGREES_TO_RADIANS * theta) * radius;
                int deltaY = sin(DEGREES_TO_RADIANS * theta) * radius;

                int imageIndex = (row + deltaY) * imageColumns + (column + deltaX);

                int pixelValue = srcImage[imageIndex];
                if (pixelValue < 10)
                {

                    accumulator++;
                }
            }

            atomicAdd(&rTable[(radius * imageColumns * imageRows) + row * imageColumns + column], accumulator);
        }
    }
}

// Did not take into account the amount of memory available on the system.
__global__ void hough_transform_kernel_shared_local_accumulator(uchar *srcImage, unsigned int *rTable, int imageRows, int imageColumns, int minimumRadius, int maximumRadius, int distance)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int column = threadIdx.x + blockIdx.x * blockDim.x;

    extern __shared__ uchar *sharedSrcImage;

    uchar *sharedSrcImageData = sharedSrcImage;

    if (row < imageRows && column < imageColumns)
    {
        sharedSrcImage[row * imageColumns + column] = srcImage[row * imageColumns + column];
    }

    __syncthreads();

    // Check if the thread is within image bounds.
    if (row < (imageRows - minimumRadius) && row > minimumRadius && column < (imageColumns - minimumRadius) && column > minimumRadius)
    {

        for (int radius = minimumRadius; radius < maximumRadius - minimumRadius; radius++)
        {
            int threshold = ((log10f(radius * 2 / 3)) * 80) / log10f(3);
            int accumulator = 0;

            for (int theta = 0; theta < 360; theta++)
            {
                int deltaX = cos(DEGREES_TO_RADIANS * theta) * radius;
                int deltaY = sin(DEGREES_TO_RADIANS * theta) * radius;

                int imageIndex = (row + deltaY) * imageColumns + (column + deltaX);

                // int pixelValue = srcImage[imageIndex];
                int pixelValue = sharedSrcImageData[imageIndex];

                if (pixelValue < 10)
                {

                    accumulator++;
                }
            }

            atomicAdd(&rTable[(radius * imageColumns * imageRows) + row * imageColumns + column], accumulator);
        }
    }
}


// Change the algorithm so that each thread focuses on one part of the image.
__global__ void hough_transform_kernel_shared_local_accumulator(uchar *srcImage, unsigned int *rTable, int imageRows, int imageColumns, int minimumRadius, int maximumRadius, int distance)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int column = threadIdx.x + blockIdx.x * blockDim.x;

    extern __shared__ uchar *sharedSrcImage;

    uchar *sharedSrcImageData = sharedSrcImage;

    if (row < imageRows && column < imageColumns)
    {
        sharedSrcImage[row * imageColumns + column] = srcImage[row * imageColumns + column];
    }

    __syncthreads();

    // Check if the thread is within image bounds.
    if (row < (imageRows - minimumRadius) && row > minimumRadius && column < (imageColumns - minimumRadius) && column > minimumRadius)
    {

        for (int radius = minimumRadius; radius < maximumRadius - minimumRadius; radius++)
        {
            int threshold = ((log10f(radius * 2 / 3)) * 80) / log10f(3);
            int accumulator = 0;

            for (int theta = 0; theta < 360; theta++)
            {
                int deltaX = cos(DEGREES_TO_RADIANS * theta) * radius;
                int deltaY = sin(DEGREES_TO_RADIANS * theta) * radius;

                int imageIndex = (row + deltaY) * imageColumns + (column + deltaX);

                // int pixelValue = srcImage[imageIndex];
                int pixelValue = sharedSrcImageData[imageIndex];

                if (pixelValue < 10)
                {

                    accumulator++;
                }
            }

            atomicAdd(&rTable[(radius * imageColumns * imageRows) + row * imageColumns + column], accumulator);
        }
    }
}

void parseRTable(std::vector<cv::Vec3f> &circles, unsigned int *rTable, int minimumRadius, int maximumRadius, int imageRows, int imageColumns)
{

    for (int row = minimumRadius; row < imageRows - minimumRadius; row++)
    {
        for (int column = minimumRadius; column < imageColumns - minimumRadius; column++)
        {
            for (int radius = 0; radius < (maximumRadius - minimumRadius); radius++)
            {
                // Check if the image at that coordinate is greater than the threshold.
                // If so, then append the circle to the circles vector.
                int rValue = rTable[(radius * imageColumns * imageRows) + row * imageColumns + column];
                if (rValue > 325)
                {

                    // printf("Adding Circle at (row, column, radius) | (%d, %d, %d) | RValue: %d\n", row, column, radius, rValue);

                    circles.push_back({(float)(column), (float)(row), (float)(radius)});

                    // circles.push_back({(float)(row + minimumRadius), (float)(column + minimumRadius), (float)(radius + minimumRadius)});
                }
            }
        }
    }
}

void houghTransform(cv::Mat &srcImage, std::vector<cv::Vec3f> &circles, int method)
{
    /* Parameters Until we make them modular */
    int distance = 1;
    int minimumRadius = 18;  // 20 is the min for the test image.
    int maximumRadius = 120; // 100 is max radius for the test image.
    int imageRows = srcImage.rows;
    int imageColumns = srcImage.cols;

    // Run Hough Transform on the CPU and return.
    if (method == 0)
    {

        hipEvent_t cpuStart, cpuStop;
        hipEventCreate(&cpuStart);
        hipEventCreate(&cpuStop);

        hipEventRecord(cpuStart);

        std::cout << "Executing Hough Transform on the CPU" << std::endl;
        cpuKernelHoughTransform(srcImage, circles, distance, minimumRadius, maximumRadius);

        hipEventRecord(cpuStop);
        hipEventSynchronize(cpuStop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, cpuStart, cpuStop);

        std::cout << "Execution Time on the CPU: " << milliseconds << std::endl;
        return;
    }

    // Allocate GPU Memory Initialize pointer for the GPU memory
    uchar *gpuImageBuffer;
    hipError_t err = hipMalloc((void **)&gpuImageBuffer, imageRows * imageColumns * sizeof(uchar));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Allocating an r-table to populate parameters for each shape.
    // Allocate the R table on the GPU.
    unsigned int *deviceRTable;
    err = hipMalloc((void **)&deviceRTable, imageColumns * imageRows * (maximumRadius - minimumRadius) * sizeof(unsigned int));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemset(deviceRTable, 0, imageColumns * imageRows * (maximumRadius - minimumRadius) * sizeof(unsigned int));
    // Copy Data from host to Device
    err = hipMemcpy(gpuImageBuffer, srcImage.ptr<uchar>(0, 0), imageRows * imageColumns * sizeof(uchar), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    std::cout << "Cuda Memory Allocated." << std::endl;

    /*
     *
     * Execute the hough transform on the specified kernel, populating the accumulator table.
     *
     */
    if (method == HOUGH_TRANSFORM_NAIVE_KERNEL)
    {
        hipEvent_t startNaiveHoughTransform, stopNaiveHoughTransform;
        hipEventCreate(&startNaiveHoughTransform);
        hipEventCreate(&stopNaiveHoughTransform);

        hipEventRecord(startNaiveHoughTransform);

        // std::cout << "Executing Hough Transform on the Naive Kernel" << std::endl;

        dim3 mygrid(ceil(imageColumns / (BLOCK_DIMENSIONS * 1.0)), ceil(imageRows / (BLOCK_DIMENSIONS * 1.0)));
        dim3 myblock(BLOCK_DIMENSIONS, BLOCK_DIMENSIONS);

        hough_transform_kernel_naive<<<mygrid, myblock>>>(gpuImageBuffer, deviceRTable, imageRows, imageColumns, minimumRadius, maximumRadius, distance);

        // std::cout << "Hough Transform Naive Kernel Execution Complete" << std::endl;
        hipEventRecord(stopNaiveHoughTransform);
        hipEventSynchronize(stopNaiveHoughTransform);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, startNaiveHoughTransform, stopNaiveHoughTransform);

        std::cout << "Execution Time on the GPU: " << milliseconds << std::endl;
    }
    else if (method == HOUGH_TRANSFORM_NAIVE_KERNEL2)
    {
    naiveKernel2:
        std::cout << "Executing Hough Transform on the Local Accumulator Kernel 2" << std::endl;
        hipEvent_t startNaiveHoughTransform, stopNaiveHoughTransform;
        hipEventCreate(&startNaiveHoughTransform);
        hipEventCreate(&stopNaiveHoughTransform);

        hipEventRecord(startNaiveHoughTransform);

        dim3 mygrid(ceil(imageColumns / (BLOCK_DIMENSIONS * 1.0)), ceil(imageRows / (BLOCK_DIMENSIONS * 1.0)));
        dim3 myblock(BLOCK_DIMENSIONS, BLOCK_DIMENSIONS);

        hough_transform_kernel_naive<<<mygrid, myblock>>>(gpuImageBuffer, deviceRTable, imageRows, imageColumns, minimumRadius, maximumRadius, distance);

        // std::cout << "Hough Transform Naive Kernel Execution Complete" << std::endl;
        hipEventRecord(stopNaiveHoughTransform);
        hipEventSynchronize(stopNaiveHoughTransform);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, startNaiveHoughTransform, stopNaiveHoughTransform);

        std::cout << "Execution Time on the GPU: " << milliseconds << std::endl;
    }
    else if (method == HOUGH_TRANSFORM_SHARED_LOCAL_ACCUMULATOR)
    {
        std::cout << "Executing Hough Transform on the Shared Memory Kernel." << std::endl;
        if (sizeof(uchar) * imageRows * imageColumns > 64000 / 4.0)
        {
            printf("The image is too large to run using shared memory. Running on global memory kernel. Rows: %d, Columns: %d\n", imageRows, imageColumns);
            exit(-1);
        }
        hipEvent_t startNaiveHoughTransform, stopNaiveHoughTransform;
        hipEventCreate(&startNaiveHoughTransform);
        hipEventCreate(&stopNaiveHoughTransform);

        hipEventRecord(startNaiveHoughTransform);

        dim3 mygrid(ceil(imageColumns / (BLOCK_DIMENSIONS * 1.0)), ceil(imageRows / (BLOCK_DIMENSIONS * 1.0)));
        dim3 myblock(BLOCK_DIMENSIONS, BLOCK_DIMENSIONS);

        hough_transform_kernel_naive<<<mygrid, myblock, sizeof(uchar) * imageRows * imageColumns>>>(gpuImageBuffer, deviceRTable, imageRows, imageColumns, minimumRadius, maximumRadius, distance);

        // std::cout << "Hough Transform Naive Kernel Execution Complete" << std::endl;
        hipEventRecord(stopNaiveHoughTransform);
        hipEventSynchronize(stopNaiveHoughTransform);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, startNaiveHoughTransform, stopNaiveHoughTransform);

        std::cout << "Execution Time on the GPU: " << milliseconds << std::endl;
    }
    else
    {
        std::cout << "Invalid Kernel Method Chosen. | (method): " << method << std::endl;
    }

    /* Modications are not being made to the image, so no copy back to the host is required. */
    // Copy data from device to host
    unsigned int *hostRTable;
    hostRTable = (unsigned int *)malloc(imageColumns * imageRows * (maximumRadius - minimumRadius) * sizeof(unsigned int));
    err = hipMemcpy(hostRTable, deviceRTable, imageColumns * imageRows * (maximumRadius - minimumRadius) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        hipFree(gpuImageBuffer);
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    parseRTable(circles, hostRTable, minimumRadius, maximumRadius, imageRows, imageColumns);

    // Free allocated memory
    hipFree(deviceRTable);
    hipFree(gpuImageBuffer);
    free(hostRTable);
    std::cout << "Cuda Memory Freed" << std::endl;

    std::cout << "GPU Hough Transform Execution Complete" << std::endl;
}

