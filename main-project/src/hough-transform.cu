#include "hip/hip_runtime.h"
#include "hough-transform.hpp"
#include <iostream>
#include <opencv2/opencv.hpp>

__global__ void hough_transform_kernel_naive(uchar *inputImage, int height, int width)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int column = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < height && column < width)
    {
        // printf("%d\n", inputImage[row * width + column]);

        inputImage[row * width + column] = 128;

        // printf("%d\n", inputImage[row * width + column]);
    }
}

void cudaHoughTransform(cv::Mat &grayscaleInputImage, cv::InputArray circles)
{

    int imageRows = grayscaleInputImage.rows;
    int imageColumns = grayscaleInputImage.cols;

    // Initializing pointers for the GPU memory
    uchar *gpuImageBuffer;

    // Error detection
    hipError_t err;

    // Allocate GPU Memory
    err = hipMalloc((void **)&gpuImageBuffer, imageRows * imageColumns * sizeof(uchar));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Copy Data from host to Device
    err = hipMemcpy(gpuImageBuffer, grayscaleInputImage.ptr<uchar>(0, 0), imageRows * imageColumns * sizeof(uchar), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    // std::cout << "Cuda Memory Allocated." << std::endl;

    // Execute kernel
    const int threads = 32;
    dim3 mygrid(ceil(imageColumns / (threads * 1.0)), ceil(imageRows / (threads * 1.0)));
    dim3 myblock(threads, threads);

    hough_transform_kernel_naive<<<mygrid, myblock>>>(gpuImageBuffer, imageRows, imageColumns);

    // Copy data from device to host
    err = hipMemcpy(grayscaleInputImage.ptr<uchar>(0, 0), gpuImageBuffer, imageRows * imageColumns * sizeof(uchar), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        hipFree(gpuImageBuffer);
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Free allocated memory
    hipFree(gpuImageBuffer);
}

__global__ void add_kernel_basic(int size, int *input1, int *input2)
{

    int thread = threadIdx.x + blockIdx.x * blockDim.x;

    if (thread < size)
    {
        input1[thread] = input1[thread] + input2[thread];
    }
}

void cudaAddKernel(int array_size, int *array_1, int *array_2)
{

    std::cout << "Running Kernel Wrapper" << std::endl;

    // Initializing pointers to the gpu memory
    int *gpu_array_1;
    int *gput_array_2;

    // allocate memory on device, check for failure
    if (hipMalloc((void **)&gpu_array_1, array_size * sizeof(int)) != hipSuccess)
    {
        std::cout << "malloc error for gpuInput1" << std::endl;
    }
    if (hipMalloc((void **)&gput_array_2, array_size * sizeof(int)) != hipSuccess)
    {
        std::cout << "malloc error for gpuInput2" << std::endl;
    }

    // copy data to device, check for failure, free device if needed

    hipError_t err; // Use this whenever calling hipMalloc and hipMemcpy.

    err = hipMemcpy(gpu_array_1, array_1, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(gput_array_2, array_2, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Execute kernel.
    dim3 mygrid(ceil(array_size / 256.0));
    dim3 myblock(256);

    add_kernel_basic<<<mygrid, myblock>>>(array_size, gpu_array_1, gput_array_2);

    // copy data to host, check for failure, free device if needed
    if (hipMemcpy(array_1, gpu_array_1, array_size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(gpu_array_1);
        hipFree(gput_array_2);
        printf("data transfer error from device to host on input1\n");
    }
    if (hipMemcpy(array_2, gput_array_2, array_size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(gpu_array_1);
        hipFree(gput_array_2);
        printf("data transfer error from device to host on input2\n");
    }

    hipFree(gpu_array_1);
    hipFree(gput_array_2);

    std::cout << "Finished Kernel Wrapper execution" << std::endl;
}
